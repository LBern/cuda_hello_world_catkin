#include "hip/hip_runtime.h"
#include "cuda_hello_world/hello-world.h"
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

const int N = 7;
const int blocksize = 7;

static __global__ void hello(char *a, int *b) {
  a[threadIdx.x] += b[threadIdx.x];
}

std::string hello_world_cu() {
  char a[N] = "Hello ";
  int b[N] = {15, 10, 6, 0, -11, 1, 0};
  std::string result = std::string(a);

  char *ad;
  int *bd;
  const int csize = N*sizeof(char);
  const int isize = N*sizeof(int);

  hipMalloc(reinterpret_cast<void**>(&ad), csize);
  hipMalloc(reinterpret_cast<void**>(&bd), isize);
  hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
  hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);

  dim3 dimBlock(blocksize, 1);
  dim3 dimGrid(1, 1);
  hello<<<dimGrid, dimBlock>>>(ad, bd);
  hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost);
  hipFree(ad);

  return result + std::string(a);
}
